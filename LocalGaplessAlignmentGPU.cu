#include "hip/hip_runtime.h"
// remember!
#include <bits/stdc++.h>
#include "Utils.h"
#include "ScoreMatrix.h"

using namespace std;

#define MAX_LEN 1024
#define debug(A) cout << #A << ": " << A << endl

// #define DEBUG_REDUCE
// #define DEBUG_KERNEL
// #define DEBUG
#define SHOW_KERNEL_CONF
#ifdef DEBUG
    #define SHOW_ALIGNMENT_SCORES
#endif

// #define REDUCE_ON_COLUMNS
#define REDUCE_ALIGNMENT_RESULT
// #define USE_LOCK

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// the type doesn't actually matter!
extern __shared__ int shared_memory[];

__constant__ byte_type _aa2num['Z' - 'A'];
__constant__ byte_type _score_matrix[ALPH_SIZE * ALPH_SIZE];

__device__ void lock(int* mutex) {
    while (atomicCAS(mutex, 0, 1) != 0) {}
}

__device__ void unlock(int* mutex) {
    atomicExch(mutex, 0);
}

void allcoate_strings_on_device_flattened(vector<string> strings, char ** d_fstr_addr, int ** d_fids_addr, int num_strs) {
    string flat_temp; int* flat_ids; char * flat_str;
    flat_ids = (int *) malloc((num_strs + 1) * sizeof(int));
    int cur_ptr = 0;
    // todo: maybe implementation of flattening can be more efficient but it's more readable now, improve it later on?
    for (int i = 0; i < strings.size(); i++) {
        flat_ids[i] = cur_ptr;
        flat_temp += strings[i];
        cur_ptr += strings[i].size();
    }
    flat_ids[num_strs] = cur_ptr;
    int total_chars_num = cur_ptr;

    flat_str = (char *) flat_temp.c_str();
    hipMalloc(d_fstr_addr, total_chars_num * sizeof(char));
    hipMalloc(d_fids_addr, (num_strs + 1) * sizeof(int));
    hipMemcpy(*d_fstr_addr, flat_str, total_chars_num * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(*d_fids_addr, flat_ids, (num_strs + 1) * sizeof(int), hipMemcpyHostToDevice);
}

// maybe a redundant function!
void free_strings_on_device_flattened(char* d_fstr, int* d_fids) {
    hipFree(d_fstr);
    hipFree(d_fids);
}

void allocate_strings_on_device_ptr2ptr(vector<string> strings, char *** d_str_ptrs_addr, char *** d_temp_strs_addr, int num_strs) {
    hipMalloc(d_str_ptrs_addr, num_strs * sizeof(char *));
    *d_temp_strs_addr = (char **) malloc(num_strs * sizeof(char*));
    char** d_str_ptrs = *d_str_ptrs_addr;
    char** d_temp_strs = *d_temp_strs_addr;
    for (int i = 0; i < num_strs; i++) {
        int q_i_len = strings[i].size();
        hipMalloc(&d_temp_strs[i],  q_i_len * sizeof(char));
        hipMemcpy(d_temp_strs[i], strings[i].c_str(), q_i_len * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(d_str_ptrs + i, &d_temp_strs[i], sizeof(char *), hipMemcpyHostToDevice);
    }
}

void free_strings_on_device_ptr2ptr(char** d_str_ptrs, char** d_temp_strs, int num_strs) {
    hipFree(d_str_ptrs);
    for (int i = 0; i < num_strs; i++) {
        hipFree(d_temp_strs[i]);
    }
}

__global__ void local_ungapped_alignment(
    int aa2num_len,
    int score_matrix_len,
    int rows_memory_len,
    int q_tmax_len,
    int opt_cells_size,
    char* target_flat_str,
    int* flat_ids,
    char* query,
    // char* query_idx,
    int q_len,
    int_type* best_scores
) {

    // assumptions:
    // - the 3rd parameter of kernel call is provided (for dynamic allocation of shared memory)
    // - the block size (or number of threads) equals query size ([tnum = q_len = bsize] in code)
    // - the target is spreaded on the row (axis=0) and the query on columns (axis=1)

    __shared__ int best_overall_score; best_overall_score = 0;
#if !defined REDUCE_ALIGNMENT_RESULT && defined USE_LOCK
    __shared__ int mutex; mutex = 0;
#endif
    // some preprocessing of variables (these would go on registers probably)
    int bid = blockIdx.x; int tid =  threadIdx.x; int bsize = blockDim.x; int tnum = bsize;
    int t_start_idx = flat_ids[bid];
    int t_len = flat_ids[bid + 1] - t_start_idx;

    // dynamically divide the whole shared memory for the shared memory variables
    byte_type* aa2num = (byte_type *)shared_memory;
    byte_type* score_matrix_flat = (byte_type *)&aa2num[aa2num_len];
    int_type* last_row = (int_type *)&score_matrix_flat[score_matrix_len];
    int_type* current_row = (int_type *)&last_row[q_len + 1];
    /* Note1: the order of these allocations actually matters and if not considered, the "misaligned address" error could occur
        for example cuda needs that int* pointers, point to addresses aligned with 32 bytes and if we allocate the char* before int* it's probable this error occurs
       Note2: the size of variables being used here is their actual size not the max size we had to specify for the kernel call
       todo: would be nice if we could free or release the additional space previously reserved but not used here */
    opt_cell* best_cells = (opt_cell*)&current_row[q_len + 1];
    char* q_cache = (char *)&best_cells[opt_cells_size];
    char* t_cache = (char *)&q_cache[q_len];
    
    // todo: extract a function for this repeated allocations if possible
    // initialize and retrieve content of shared memory variables
    for (int i = 0; i < (aa2num_len + tnum - 1) / tnum; i++) {
        int idx = i * tnum + tid;
        if (idx < aa2num_len) {
            aa2num[idx] = _aa2num[idx];
        }
    }

    for (int i = 0; i < (score_matrix_len + tnum - 1) / tnum; i++) {
        int idx = i * tnum + tid;
        if (idx < score_matrix_len) {
            score_matrix_flat[idx] = _score_matrix[idx];
        }
    }

    // Note: thread with index i works on column i+1 of alignment matrix and need column i to compute it
    last_row[tid] = 0;
    // current row does not need initialization!
    
    q_cache[tid] = query[tid];
    // q_cache[tid] = query_idx[tid];

    for (int i = 0; i < (t_len + tnum - 1) / tnum; i++) {
        int idx = i * tnum + tid;
        if (idx < t_len) {
            t_cache[idx] = target_flat_str[idx + t_start_idx];
        }
    }

#ifdef REDUCE_ON_COLUMNS
    best_cells[tid].score = 0;
    best_cells[tid].diagonal_idx = USHRT_MAX;
#else
    int diagonal_size = t_len + q_len - 1;
    for (int i = 0; i < (diagonal_size + tnum - 1) / tnum; i++) {
        int idx = i * tnum + tid;
        if (idx < diagonal_size) {
            best_cells[idx].score = 0;
            best_cells[idx].diagonal_idx = idx;
        }
    }
#endif

    __syncthreads();

    // now actual alignment algorithm can begin :D
    // todo: test the thrust and cuBLAS library and the reduce method for acquiring extermum in an array
    for (int row = 1; row <= t_len; row++) {
        // this thread works on column = tid + 1 and q[tid] char
        // int char_idx = (int) q_cache[tid];
        // int mat_idx = char_idx * ALPH_SIZE + aa2num[int(t_cache[row-1] - 'A')];
        int mat_idx = aa2num[(q_cache[tid] - 'A')] * ALPH_SIZE + aa2num[(t_cache[row-1] - 'A')];
        byte_type substitution_score = score_matrix_flat[mat_idx];
#ifdef DEBUG_KERNEL
        printf("(r%d%c, c%2d%c)%2d\n", row, t_cache[row-1], tid+1, q_cache[tid], substitution_score);
#endif
        int_type current_score = max(0, last_row[tid] + substitution_score);
        int_type current_diagonal = row - tid + q_len - 2; // row - col + q_len - 1
        current_row[tid+1] = current_score; // there is no race condition here!

        // each thread works on different diagonal or column (considering columns of threads differ and rows are the same)
        // so there is no race condition here yet either
#ifdef REDUCE_ON_COLUMNS
        if (current_score > best_cells[tid].score) {
            best_cells[tid].score = current_score;
            best_cells[tid].diagonal_idx = current_diagonal;
#else
        if (current_score > best_cells[current_diagonal].score) {
            best_cells[current_diagonal].score = current_score;
#endif
            // here we have a race condition to update the best_overall_score and best_diagonal
#if !defined REDUCE_ALIGNMENT_RESULT && defined USE_LOCK
            while (atomicCAS(&mutex, 0, 1) != 0); // lock
            if (current_score > best_overall_score) {
                best_overall_score = current_score;
                // best_diag_idx = current_diagonal;
            }
            atomicExch(&mutex, 0); // unlock
#elif !defined REDUCE_ALIGNMENT_RESULT && !defined USE_LOCK
            atomicMax(&best_overall_score, current_score);
#endif
        }
        // to reassure that all current_row cells are updated before using them
        __syncthreads();
#ifdef DEBUG_KERNEL
        printf("diag%2d(r%d, c%2d)%2d\n", current_diagonal, row, tid+1, current_score);
#endif
        // last_row[0] is already zero and there is no need to update the value (btw same can be said for last_row[q_len] considering it will not be used but anyway!)
        last_row[tid+1] = current_row[tid+1];
        __syncthreads();
    }

#ifdef REDUCE_ALIGNMENT_RESULT
    #if defined DEBUG_REDUCE & !defined REDUCE_ON_COLUMNS
    if (!tid) {
        printf("before reduction:\n");
        for (int i = 0; i < diagonal_size; i++) {
            printf("score:%d, idx:%d\n", best_cells[i].score, best_cells[i].diagonal_idx);
        }
    }
    #endif

    #if !defined REDUCE_ON_COLUMNS
    // this whole reduction on diagonals complexity time is not o(logn) but it's o(Lt/Lq) + o(logn)
    for (int i = 0; i < (diagonal_size + tnum - 1) / tnum; i++) {
        int idx = i * tnum + tid;
        if (idx < diagonal_size && best_cells[tid].score < best_cells[idx].score) {
            best_cells[tid].score = best_cells[idx].score;
            best_cells[tid].diagonal_idx = best_cells[idx].diagonal_idx;
        }
    }
    #endif

    for (int i = (tnum + 1) / 2; i > 0; i = (i+1) / 2) {
        int idx = tid + i;
        if (tid < i && idx < tnum) {
    #ifdef DEBUG_REDUCE
            printf("(tid:%d, idx:%d| cur_size:%d block_size:%d)\n", tid, idx, i, tnum);
    #endif
            if (best_cells[tid].score < best_cells[idx].score) {
                best_cells[tid].score = best_cells[idx].score;
                best_cells[tid].diagonal_idx = best_cells[idx].diagonal_idx;
            }
        }
        __syncthreads();
    #ifdef DEBUG_REDUCE
        if (!tid) for (int j = 0; j < i; j++) {
            printf("score:%d, idx:%d\n", best_cells[j].score, best_cells[j].diagonal_idx);
        }
    #endif
        if (i == 1)
            break;
    }
    // now best_cells[0] holds the maximum best score
    best_overall_score = best_cells[0].score;
    #ifdef DEBUG_REDUCE
        printf("end of reducing from thread %d --> best_diagonal:%d, best_socore:%d\n", tid, best_cells[0].score, best_cells[0].diagonal_idx);
    #endif
#else
    #ifdef DEBUG_KERNEL
    printf("end of thread %d, best_overall_score: %d\n", tid, best_overall_score);
    #endif
#endif

    if (!tid) {
#if defined REDUCE_ALIGNMENT_RESULT && defined DEBUG_REDUCE
        printf("%d, bid:%d, best_overall_score:%d, best_diag: %d\n", t_len, bid, best_overall_score, best_cells[0].diagonal_idx);
#endif
        best_scores[bid] = best_overall_score;
    }
}

__global__ void local_ungapped_alignment_on_diagonal(
    int aa2num_len,
    int score_matrix_len,
    int q_tmax_len,
#ifdef REDUCE_ALIGNMENT_RESULT
    int opt_cells_size,
#endif
    char* target_flat_str,
    int* flat_ids,
    char* query,
    int q_len,
    int_type* best_scores
) {

    // assumptions:
    // - the 3rd parameter of kernel call is provided (for dynamic allocation of shared memory)
    // - the block size (or number of threads) equals mx=max(Lq, max{Lt})
    // - the target is spreaded on the row (axis=0) and the query on columns (axis=1)
    // - thread with "tid" works on diagonal_idx of tid and maybe mx + tid
    // - starting cell of diagonal_idx "did" is (row=max(1, did - Lq + 2), col=max(1, Lq - did)) [reminder: row-> [1, Lt], col-> [1, Lq], diag_idx-> [0, Lt+Lq-1)]
    __shared__ int best_overall_score; best_overall_score = 0;
    // some preprocessing of variables (these would go on registers probably)
    int bid = blockIdx.x; int tid =  threadIdx.x; int bsize = blockDim.x; int tnum = bsize;
    int t_start_idx = flat_ids[bid];
    int t_len = flat_ids[bid + 1] - t_start_idx;
    int diagonal_size = t_len + q_len - 1;
    int actual_tnum = max(t_len, q_len);

    // dynamically divide the whole shared memory for the shared memory variables
    byte_type* aa2num = (byte_type *)shared_memory;
    byte_type* score_matrix_flat = (byte_type *)&aa2num[aa2num_len];
#ifdef REDUCE_ALIGNMENT_RESULT
    opt_cell* best_cells = (opt_cell*)&score_matrix_flat[score_matrix_len];
    char* q_cache = (char *)&best_cells[opt_cells_size];
    char* t_cache = (char *)&q_cache[q_len];
#else
    char* q_cache = (char *)&score_matrix_flat[score_matrix_len];
    char* t_cache = (char *)&q_cache[q_len];
#endif

    // todo: extract a function for this repeated allocations if possible
    // initialize and retrieve content of shared memory variables
    for (int i = 0; i < (aa2num_len + tnum - 1) / tnum; i++) {
        int idx = i * tnum + tid;
        if (idx < aa2num_len) {
            aa2num[idx] = _aa2num[idx];
        }
    }

    for (int i = 0; i < (score_matrix_len + tnum - 1) / tnum; i++) {
        int idx = i * tnum + tid;
        if (idx < score_matrix_len) {
            score_matrix_flat[idx] = _score_matrix[idx];
        }
    }

    q_cache[min(tid, q_len-1)] = query[min(tid, q_len-1)];
    t_cache[min(tid, t_len-1)] = target_flat_str[min(tid, t_len-1) + t_start_idx];

    __syncthreads();

    // now actual alignment algorithm can begin :D
    int diag_idx = tid;
    bool best_diag_loc_first = true;
    int_type best_score = 0;
    int_type current_score = 0;
    if (tid < actual_tnum) {
        for (int row = max(1, diag_idx - q_len + 2), col = max(1, q_len - diag_idx); row <= t_len && col <= q_len; row++, col++) {
            int mat_idx = aa2num[(q_cache[col-1] - 'A')] * ALPH_SIZE + aa2num[(t_cache[row-1] - 'A')];
            byte_type substitution_score = score_matrix_flat[mat_idx];
            current_score = max(0, current_score + substitution_score);
            if (current_score > best_score) {
                best_score = current_score;
            }
        }
    }
    if (tid + actual_tnum < diagonal_size) {
        diag_idx = tid + actual_tnum;
        current_score = 0;
        for (int row = max(1, diag_idx - q_len + 2), col = max(1, q_len - diag_idx); row <= t_len && col <= q_len; row++, col++) {
            int mat_idx = aa2num[(q_cache[col-1] - 'A')] * ALPH_SIZE + aa2num[(t_cache[row-1] - 'A')];
            byte_type substitution_score = score_matrix_flat[mat_idx];
            current_score = max(0, current_score + substitution_score);
            if (current_score > best_score) {
                best_score = current_score;
                best_diag_loc_first = false;
            }
        }
    }

#ifndef REDUCE_ALIGNMENT_RESULT
    atomicMax(&best_overall_score, best_score);
    __syncthreads();
#else
    if (tid < actual_tnum) {
        best_cells[tid].score = best_score;
        best_cells[tid].diagonal_idx = best_diag_loc_first * tid + !best_diag_loc_first * (tid + actual_tnum);
    }
    __syncthreads();
    #if defined DEBUG_REDUCE
    if (!tid) {
        for (int i = 0; i < actual_tnum; i++) {
            printf("i:%d, score:%d, idx:%d\n", i, best_cells[i].score, best_cells[i].diagonal_idx);
        }
    }
    #endif

    for (int i = (actual_tnum + 1) / 2; i > 0; i = (i+1) / 2) {
        int idx = tid + i;
        if (tid < i && idx < actual_tnum) {
    #ifdef DEBUG_REDUCE
            printf("(tid:%d, idx:%d| cur_size:%d block_size:%d)\n", tid, idx, i, tnum);
    #endif
            if (best_cells[tid].score < best_cells[idx].score) {
                best_cells[tid].score = best_cells[idx].score;
                best_cells[tid].diagonal_idx = best_cells[idx].diagonal_idx;
            }
        }
        __syncthreads();
    #ifdef DEBUG_REDUCE
        if (!tid) for (int j = 0; j < i; j++) {
            printf("score:%d, idx:%d\n", best_cells[j].score, best_cells[j].diagonal_idx);
        }
    #endif
        if (i == 1)
            break;
    }

    // now best_cells[0] holds the maximum best score
    best_overall_score = best_cells[0].score;

#endif

    if (!tid) {
#if defined REDUCE_ALIGNMENT_RESULT && defined DEBUG_REDUCE
        printf("%d, bid:%d, best_score:%d, best_diag: %d\n", t_len, bid, best_score, best_cells[0].diagonal_idx);
#endif
        best_scores[bid] = best_overall_score;
    }
}

void call_kernel(string query, vector<string>& targets, bool on_columns=true) {

/* some todos for more efficiency:
    - merge minor memory transactions
    - use pinned memory
    - clean up more if possible!
*/
    int max_target_len = 0;
    int sum_target_len = 0;
    for (auto &t: targets) {
        int temp_len = t.size();
        max_target_len = max(max_target_len, temp_len);
        sum_target_len += temp_len;
    }
    char * q_str = (char *) query.c_str();
    int q_len = query.size();
    // convert query to its indexes on aa2num
    // char * q_str_idx = (char *)malloc(q_len * sizeof(char));
    // for (int i = 0; i < q_len; i++) {
    //     q_str_idx[i] = (char)aa2num[q_str[i] - 'A'];
    // }
    int num_target_strs = targets.size();

    // allocate and initialize some necessary variables on device for kernel call
    char * d_query_str; char * d_targets_fstr; int * d_target_indices; int_type *best_scores;
    hipMalloc(&d_query_str, q_len * sizeof(char));
    hipMemcpy(d_query_str, q_str, q_len * sizeof(char), hipMemcpyHostToDevice);

    // char * d_query_str_idx; char * d_targets_fstr; int * d_target_indices; int *best_scores;
    // hipMalloc(&d_query_str_idx, q_len * sizeof(char));
    // hipMemcpy(d_query_str_idx, q_str_idx, q_len * sizeof(char), hipMemcpyHostToDevice);
    allcoate_strings_on_device_flattened(targets, &d_targets_fstr, &d_target_indices, num_target_strs);
    hipMallocManaged(&best_scores, num_target_strs * sizeof(int_type));
    // be sure to have called init_score_matrix() before using score matrix
    // todo: do sth about this init_score_matrix necessity
    hipMemcpyToSymbol(HIP_SYMBOL(_score_matrix), score_matrix_flattened, SCORE_MATRIX_SIZE * sizeof(byte_type));
    hipMemcpyToSymbol(HIP_SYMBOL(_aa2num), aa2num, ('Z' - 'A') * sizeof(byte_type));

    // calculating the (maximum) size of different variables which will go on shared-memory
    int aa2num_len = 'Z' - 'A';
    int score_matrix_len = SCORE_MATRIX_SIZE;
    int rows_memory_len = 2 * (q_len + 1); // current row + last row
    int q_tmax_len = (max_target_len + q_len);
#ifdef REDUCE_ON_COLUMNS
    int opt_cells_size = q_len; // column_size
#else
    int max_diagonal_size = (max_target_len + q_len - 1);
    int opt_cells_size = max_diagonal_size;
#endif
    // calculating the maximum needed shared-memory size in bytes
    int shared_memory_size;
    if (on_columns) {
        shared_memory_size = aa2num_len * sizeof(byte_type)
        + score_matrix_len  * sizeof(byte_type)
        + rows_memory_len * sizeof(int_type)
        + q_tmax_len  * sizeof(char)
        + opt_cells_size * sizeof(opt_cell)
        ;

        local_ungapped_alignment<<<num_target_strs, q_len, shared_memory_size>>> (
            aa2num_len,
            score_matrix_len,
            rows_memory_len,
            q_tmax_len,
            opt_cells_size,
            d_targets_fstr,
            d_target_indices,
            d_query_str,
            // d_query_str_idx,
            q_len,
            best_scores
        );
    } else {
        opt_cells_size = max(max_target_len, q_len);

        shared_memory_size = aa2num_len * sizeof(byte_type)
        + score_matrix_len  * sizeof(byte_type)
        + q_tmax_len  * sizeof(char)
#ifdef REDUCE_ALIGNMENT_RESULT
        + opt_cells_size * sizeof(opt_cell)
#endif
        ;
        local_ungapped_alignment_on_diagonal<<<num_target_strs, opt_cells_size, shared_memory_size>>> (
            aa2num_len,
            score_matrix_len,
            q_tmax_len,
#ifdef REDUCE_ALIGNMENT_RESULT
            opt_cells_size,
#endif
            d_targets_fstr,
            d_target_indices,
            d_query_str,
            q_len,
            best_scores
        );
    }
#ifdef SHOW_KERNEL_CONF
    printf("shared_memory_size: %d, num_targets=grid_size: %d, q_len: %d, max_t_len: %d, mode: %s\n", shared_memory_size, num_target_strs, q_len, max_target_len, on_columns?"on_columns":"on_diagonals");
#endif
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
#ifdef SHOW_ALIGNMENT_SCORES
    cout << "final scores:" << endl;
    for (int i = 0; i < num_target_strs; i++) {
        cout << query.substr(0, 10) << "," << targets[i].substr(0, 10);
        printf("(%4d,%4d): %4d\n", q_len, (int)targets[i].size(), best_scores[i]);
    }
#endif
    // freeing device memory
    free_strings_on_device_flattened(d_targets_fstr, d_target_indices);
    hipFree(d_query_str);
    // hipFree(d_query_str_idx);
    hipFree(best_scores);
}

void measure_kernel_time(string query, vector<string> targets, bool on_columns=true, bool verbose=true, int number_of_calls=20) {
    clock_t start_clock, end_clock;
    long maximum_clocks = 0, minimum_clocks = LLONG_MAX, sum_clocks = 0;

    for (int i = 0; i < number_of_calls; i++) {
        start_clock = clock();

        call_kernel(query, targets, on_columns);

        end_clock = clock();

        long execution_clocks = end_clock - start_clock;
        maximum_clocks = max(maximum_clocks, execution_clocks), minimum_clocks = min(minimum_clocks, execution_clocks);
        sum_clocks += execution_clocks;
        if (verbose) cout << "execution clocks: " << execution_clocks << endl; // divide by CLOCKS_PER_SEC if actual time is needed
    }
    debug(maximum_clocks); debug(minimum_clocks); cout << "avg_clocks: " << sum_clocks / number_of_calls << endl;
}

int main(int argc, char** argv) {
    vector<string> queries, targets;
    init_score_matrix();
    // arguments are optional and the default ones are targets=TestSamples/targets.txt, is_fasta=true, on_columns=true, query=TestSamples/queries.txt[0]
    // the 1s argument tells the file path, the 2nd one tells if target file is fasta, the 3rd determines alignment method (on-columns or on-diagonals) and the 4th tell query path
    // only string "0" can change 2nd and 3rd arguments to "not fasta" and "not on_column" (i.e "on_diagonal")
    string target_path = "TestSamples/targets.txt";
    string query_path = "TestSamples/queries.txt";
    bool is_target_fasta = true;
    bool on_columns = true;
    if (argc > 1) {
        target_path = argv[1];
        if (argc > 2) {
            if(string(argv[2]) == "0") is_target_fasta = false;
            if (argc > 3) {
                if (string(argv[3]) == "0") on_columns = false;
                if (argc > 4) {
                    query_path = argv[4];
                }
            }
        }
    }
    init_input_from_file(query_path, queries, false);
    init_input_from_file(target_path, targets, is_target_fasta);

#ifdef DEBUG
    call_kernel(queries[0], targets, on_columns);
#else
    measure_kernel_time(queries[0], targets, on_columns, true, 10);
#endif
    
    return 0;
}